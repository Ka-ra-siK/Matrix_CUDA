#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <math.h>
#include <time.h>
#include <locale.h>

//#define N 800
hipError_t err = hipSuccess;
//float A[N * (N + 1)];

void check_err() {
	if (err != hipSuccess) {
		fprintf(stderr, "Failed ", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void getMatrix(int N, float* A) {
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++)
			A[i + j * N] = rand() % 10;
		A[i + N * N] = 0;
	}
}

__global__ void gauss_stage1(float* a, int n, float x, int N) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i <= N - n + 1) {
		a[n + N * (i + n)] /= x;
	}
}

__global__ void gauss_stage2(float* a, int n, int i, float x, int N) {
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	if (j <= N - n - 1) {
		a[i + N * (j + n + 1)] -= a[n + N * (j + n + 1)] * x;
	}
}

__global__ void gauss_stage3(float* a, int n, int N) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n) {
		a[i + N * N] -= a[n + N * N] * a[i + n * N];
	}
}


void findDeterminant(int N) {
	check_err();

	int threadsPerBlock = 128,
		blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock,
		size = sizeof(float) * N * (N + 1);
	float *A = (float*)malloc(N * (N+1) * sizeof(float));
	getMatrix(N, A);
	float* _A = NULL;
	err = hipMalloc((void**)&_A, size); check_err();

	err = hipMemcpy(_A, A, size, hipMemcpyHostToDevice); check_err();

	for (int i = 0; i < N; i++) {
		gauss_stage1 << <blocksPerGrid, threadsPerBlock >> > (_A, i, A[i + i * N], N);
		for (int j = i + 1; j < N; j++)
			gauss_stage2 << <blocksPerGrid, threadsPerBlock >> > (_A, i, j, A[j + i * N], N);
	}

	for (int i = N - 1; i > 0; i--)
		gauss_stage3 << <blocksPerGrid, threadsPerBlock >> > (_A, i, N);
	double det = 1.0;
	for (int j = 0; j < N; j++)
		det *= A[j + N * N];
	hipFree(_A);
	free(A);
}

int main(void) {
	setlocale(LC_ALL, "C");
	srand(time(NULL));
	const int size = 1500;
	FILE* text;
	for (int N = 2; N <= size; N++) {
	srand(time(NULL));
	int before = clock();
	findDeterminant(N);
	double time = (clock() - before) / (double)CLOCKS_PER_SEC;
	printf("Time: %.3f sec.\n", time);
	text = fopen("GPU.txt", "a");
	fprintf(text, "%d %.3f\n", N, time);
	fclose(text);
	}
	system("pause");
	return 0;
}